#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"


#ifndef BLOCKSIZE				// run 16, 32, 64
#define BLOCKSIZE		32		// number of threads per block 
#endif

#ifndef SIZE
#define SIZE			16000	// 16k, 32k, 64k, 128k, 256k and 512k
#endif

//#ifndef NUMTRIALS	
//#define NUMTRIALS		10		// to make the timing more accurate
//#endif

#ifndef TOLERANCE
#define TOLERANCE		0.00001f	// tolerance to relative error
#endif

// ranges for the random numbers:
const float XCMIN =	 0.0;
const float XCMAX =	 2.0;
const float YCMIN =	 0.0;
const float YCMAX =	 2.0;
const float RMIN  =	 0.5;
const float RMAX  =	 2.0;

float Ranf( float low, float high ){
    float r = (float)rand();              // 0 - RAND_MAX
    float t = r / (float) RAND_MAX;       // 0. - 1.
    return low + t * ( high - low );
}

void TimeOfDaySeed( ){
	struct tm y2k = { 0 };
	y2k.tm_hour = 0; 
    y2k.tm_min = 0; 
    y2k.tm_sec = 0;
	y2k.tm_year = 100; 
    y2k.tm_mon = 0; 
    y2k.tm_mday = 1;
	time_t  timer;
	time( &timer );
	double seconds = difftime( timer, mktime(&y2k) );
	unsigned int seed = (unsigned int)( 1000.*seconds );    // milliseconds
	srand( seed );
}


// Monte Carlo Simulation (CUDA Kernel)
// int array C holds the total number of 'hits' per block in each index
__global__  void MonteCarlo( float *X, float *Y, float *R, int *C ){
	__shared__ int hits[BLOCKSIZE];

	unsigned int numItems = blockDim.x;
	unsigned int tnum = threadIdx.x;
	unsigned int wgNum = blockIdx.x;
	unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;

	hits[tnum] = 1;

	// solve for the intersection using the quadratic formula:	
	float a = 2.;
	float b = -2.*( X[gid] + Y[gid] );
	float c = X[gid]*X[gid] + Y[gid]*Y[gid] - R[gid]*R[gid];
	float d = b*b - 4.*a*c;

	// If d is less than 0., then the circle was completely missed. (Case A) 
	if( d < 0.){
		hits[tnum] = 0;
	}
	else{
		// else it hits the circle...
		// get the first intersection:
		d = sqrt( d );
		float t1 = (-b + d ) / ( 2.*a );	// time to intersect the circle
		float t2 = (-b - d ) / ( 2.*a );	// time to intersect the circle
		float tmin = t1 < t2 ? t1 : t2;		// only care about the first intersection

		// If tmin is less than 0., then the circle completely engulfs the laser pointer. (Case B)
		if ( tmin < 0.){
			hits[tnum] = 0;
		}
		else{
			// where does it intersect the circle?
			float xcir = tmin;
			float ycir = tmin;

			// get the unitized normal vector at the point of intersection:
			float nx = xcir - X[gid];
			float ny = ycir - Y[gid];
			float n = sqrt( nx * nx + ny * ny );
			nx /= n;	// unit vector
			ny /= n;	// unit vector

			// get the unitized incoming vector:
			float inx = xcir - 0.;
			float iny = ycir - 0.;
			float in = sqrt( inx * inx + iny * iny );
			inx /= in;	// unit vector
			iny /= in;	// unit vector

			// get the outgoing (bounced) vector:
			float dot = inx*nx + iny*ny;
			//float outx = inx - 2.*nx*dot;	// angle of reflection = angle of incidence`
			float outy = iny - 2.*ny*dot;	// angle of reflection = angle of incidence`

			// find out if it hits the infinite plate:
			float t = ( 0. - ycir ) / outy;

			// If t is less than 0., then the reflected beam went up instead of down
			if( t < 0.){
				hits[tnum] = 0;
			}
		}
	}

	// add up all hits
	for (int offset = 1; offset < numItems; offset *= 2){
		int mask = 2 * offset - 1;
		__syncthreads();
		if ((tnum & mask) == 0){
			hits[tnum] += hits[tnum + offset];
		}
	}

	__syncthreads();
	// record results to array of hits per block
	if (tnum == 0)
		C[wgNum] = hits[0];
}


// main program:
int main( int argc, char* argv[ ] ){
	int dev = findCudaDevice(argc, (const char **)argv);

	TimeOfDaySeed();

	// allocate host memory:
	float * hxcs = new float [ SIZE ];
	float * hycs = new float [ SIZE ];
	float * hrs = new float [ SIZE ];
	int * hC = new int [ SIZE/BLOCKSIZE ];

	// fill in arrays with random values in the given ranges:
	for( int i = 0; i < SIZE; i++ ){
		hxcs[i] = Ranf(XCMIN, XCMAX);
		hycs[i] = Ranf(YCMIN, YCMAX);
		hrs[i] = Ranf(RMIN, RMAX);
	}

	// allocate device memory:
	float *dxcs, *dycs, *drs;
	int *dC;
	dim3 dimsX( SIZE, 1, 1 );
	dim3 dimsY( SIZE, 1, 1 );
	dim3 dimsR( SIZE, 1, 1 );
	dim3 dimsC( SIZE/BLOCKSIZE, 1, 1 );


	hipError_t status;
	status = hipMalloc( reinterpret_cast<void **>(&dxcs), SIZE*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dycs), SIZE*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&drs), SIZE*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dC), (SIZE/BLOCKSIZE)*sizeof(int) );
		checkCudaErrors( status );


	// copy host memory to the device:
	status = hipMemcpy( dxcs, hxcs, SIZE*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( dycs, hycs, SIZE*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( drs, hrs, SIZE*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );

	// setup the execution parameters:
	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid( SIZE / threads.x, 1, 1 );

	// Create and start timer
	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:
	hipEvent_t start, stop;
	status = hipEventCreate( &start );
		checkCudaErrors( status );
	status = hipEventCreate( &stop );
		checkCudaErrors( status );

	// record the start event:
	status = hipEventRecord( start, NULL );
		checkCudaErrors( status );

	// execute the kernel:
	MonteCarlo<<< grid, threads >>>( dxcs, dycs, drs, dC);

	// record the stop event:
	status = hipEventRecord( stop, NULL );
		checkCudaErrors( status );

	// wait for the stop event to complete:
	status = hipEventSynchronize( stop );
		checkCudaErrors( status );

	float msecTotal = 0.0f;
	status = hipEventElapsedTime( &msecTotal, start, stop );
		checkCudaErrors( status );

	// compute and print the performance
	double secondsTotal = 0.001 * (double)msecTotal;
	double TrialsPerSecond = (float)SIZE / secondsTotal;
	double megaTrialsPerSecond = TrialsPerSecond / 1000000.;
	fprintf( stderr, "Blocksize = %d, NumTrials = %d, MegaTrials/Second = %10.6lf\n", BLOCKSIZE, SIZE, megaTrialsPerSecond );


	// copy result from the device to the host:
	status = hipMemcpy( hC, dC, (SIZE/BLOCKSIZE)*sizeof(float), hipMemcpyDeviceToHost );
		checkCudaErrors( status );

	// check the sum of all recordings in C:
	int sumHits = 0;
	for(int i = 0; i < SIZE/BLOCKSIZE; i++ ){
		sumHits += hC[i];
	}
	
	// probability around 42
	fprintf( stderr, "probability = %4.6lf\n", (float)sumHits/(float)SIZE);


	// clean up memory:
	delete [ ] hxcs;
	delete [ ] hycs;
	delete [ ] hrs;
	delete [ ] hC;

	status = hipFree( dxcs );
		checkCudaErrors( status );
	status = hipFree( dycs );
		checkCudaErrors( status );
	status = hipFree( drs );
		checkCudaErrors( status );
	status = hipFree( dC );
		checkCudaErrors( status );

	return 0;
}